
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void HelloThread()
{
	int block_id = blockIdx.x + blockIdx.y * blockDim.x + blockIdx.z * (blockDim.x * blockDim.y);
	int thread_id = threadIdx.x + threadIdx.y * 2 + threadIdx.z * (2 * 2);
	const int thread_amount = 2 * 2 * 2;
	int global_thread_id = block_id * thread_amount + thread_id;
	printf(
		"Hello Threads!  blk %2d@(%d,%d,%d), thr%2d@(%d,%d,%d), %3d\n",
		block_id, blockIdx.x, blockIdx.y, blockIdx.z,
		thread_id, threadIdx.x, threadIdx.y, threadIdx.z,
		global_thread_id);
}

int main()
{
	dim3 a(3, 2, 6), b(2, 2, 2); // blocks: 3*2*6, threads: 2*2*2
	HelloThread<<<a, b>>>();	 // 36 blocks, 8 threads = 288 parallel
	hipDeviceSynchronize();	 // wait CPU to finish process written above
	return 0;
}
